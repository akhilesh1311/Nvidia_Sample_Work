#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#define NUMBER 23

__global__ void uncoalesced();

__global__ void uncoalesced(int *d_A){
	int blockIdfinal,threadIdfinal;
	blockIdfinal=blockIdx.y+gridDim.y*blockIdx.x;
//	printf(" lol %d %d %d",);
	threadIdfinal=blockIdfinal*blockDim.x*blockDim.y + (threadIdx.y+threadIdx.x*blockDim.y);
	if(threadIdfinal<NUMBER){
		*(d_A+threadIdfinal)=100;
	//	printf(" %d",threadIdfinal);
	}
}

__global__ void trial(int *d_A){
	printf("this has to be fast   %d %d \n ",threadIdx.x,threadIdx.y);
	
}

int main(int argc, char *argv[]){
//	printf("\nUncoalesced accesses to the Global memory of Dram of GPU\n");
/*	dim3 block(4,4);
	dim3 thread(3,3);
	trial<<<block,thread>>>();
	hipDeviceSynchronize();*/
	
	size_t size=NUMBER*sizeof(int);
	int *d_A;
	hipMalloc(&d_A,size);
	
	dim3 block(5,4);
	dim3 thread(10,1);
	uncoalesced<<<block,thread>>>(d_A);
	
	int *h_A;
	h_A=(int *)malloc(sizeof(int)*NUMBER);
	
	hipMemcpy(h_A,d_A,size,hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	for(int i=0;i<NUMBER;i++){
		printf("  %d) %d",i,h_A[i]);
	}
	
	return 0;
}
