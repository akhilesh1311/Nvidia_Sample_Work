#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include<stdlib.h>
#define GAP 100
#define NUMBER 400
#define THREAD 37
#define BLOCK 11
#define LIMIT 500000

__global__ void uncoalesced(int *d_A,int *d_local){
	
	int threadIdfinal=blockIdx.x*blockDim.x+threadIdx.x;
	if(threadIdfinal*GAP+GAP<=NUMBER){
	for(int i=threadIdfinal*GAP;i<threadIdfinal*GAP+GAP;i++){
		if(d_local[threadIdfinal]<d_A[i]){
			d_local[threadIdfinal]=d_A[i];
		}
	}
	}else if(threadIdfinal*GAP<NUMBER){
		for(int i=threadIdfinal*GAP;i<NUMBER;i++){
		if(d_local[threadIdfinal]<d_A[i]){
			d_local[threadIdfinal]=d_A[i];
		}
	}
	}
}

__global__ void coalesced(int *d_A,int *d_local){
	int threadIdfinal=blockIdx.x*blockDim.x+threadIdx.x;
	if(threadIdfinal<NUMBER/GAP){
		for(int i=threadIdfinal;i<NUMBER;i=i+GAP){
			if(d_local[threadIdfinal]<d_A[i]){
				d_local[threadIdfinal]=d_A[i];
			}	
		}
	}
}

/*__global__ void coalesced(int *d_A,int *d_local){
	int threadIdfinal=blockIdx.x*blockDim.x+threadIdx.x;
	int localmax=0;
	if(threadIdfinal<NUMBER/GAP){
		for(int i=threadIdfinal;i<NUMBER;i=i+GAP){
			if(localmax<d_A[i]){
				localmax=d_A[i];
			}	
		}
		d_local[threadIdfinal]=localmax;
	}
}*/


int main(int argc, char *argv[]){
	size_t size=NUMBER*sizeof(int);
	size_t sizesol=(NUMBER+GAP)*sizeof(int)/GAP;
	int *d_A;
	hipMalloc(&d_A,size);

	int *h_A;
	h_A=(int *)malloc(sizeof(int)*NUMBER);
	
	time_t t;
	srand((unsigned)time(&t));
	
	for(int i=0;i<NUMBER;i++){
		h_A[i]=rand()%LIMIT;
	}
		
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	dim3 block(BLOCK);
	dim3 thread(THREAD);
	
	int *d_local;
	hipMalloc(&d_local,sizesol);
	
	int *h_B;
	h_B=(int *)malloc(sizesol);
	
	for(int i=0;i<NUMBER/GAP+1;i++){
		printf("~~%d~~",i);
		h_B[i]=0;
	}
	
	hipMemcpy(d_local,h_B,sizesol,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	coalesced<<<block,thread>>>(d_A,d_local);
	
	
	int *h_global;
	h_global=(int *)malloc(sizeof(int)*NUMBER);
	
	hipMemcpy(h_global,d_local,sizesol,hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	for(int i=0;i<NUMBER;i++){
		printf("  %d) %d\n",i,h_A[i]);
	}
	
	int global_max=0;
	for(int i=0;i<NUMBER/GAP+1;i++){
		printf("dodo   %d\n",h_global[i]);
		if(global_max<h_global[i]){
			
			global_max=h_global[i];
		}
	}
	printf("alas, here comes hte final output  %d\n",global_max);
	hipDeviceReset();
	return 0;
}
